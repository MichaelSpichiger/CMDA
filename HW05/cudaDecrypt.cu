#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

  __device__ unsigned int modP(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
  __device__ unsigned int modE(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = modP(aExpb, z, p);
    z = modP(z, z, p);
    b /= 2;
  }
  return aExpb;
}
   __global__ void kernelFindKey(unsigned int g,unsigned int p, unsigned int h, unsigned int *pointer) {
   
   int thread = threadIdx.x;
   int block = blockIdx.x;
   int Nblock = blockDim.x;

   int id = thread + block*Nblock;

   if (id<(p-1)) {
   if (modE(g,id,p)==h) {
        printf("Secret key found! x = %u \n", id);
        *pointer = id;
      } 
   }
}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

  FILE* key;
  key = fopen("bonus_public_key.txt", "r");
  fscanf(key, "%u \n %u \n %u \n %u \n", &n, &p, &g, &h);
  fclose(key);

  FILE* mess = fopen("bonus_message.txt", "r");
  fscanf(mess, "%u \n", &Nints);
  
  unsigned int *Zmessage = (unsigned int *) malloc(Nints*sizeof(unsigned int));
  unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));
 
  for (unsigned int i=0; i < Nints; i++) {
   fscanf(mess, "%u %u \n", &Zmessage[i], &a[i]);
  }
 

  fclose(mess);
  unsigned int Nthreads = 32;
  unsigned int Nblocks = ((unsigned int)(p-1)+Nthreads-1)/Nthreads;  

  unsigned int *h_k = (unsigned int *) malloc(sizeof(unsigned int));
  unsigned int *d_k;

  // find the secret key
  if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    hipMalloc(&d_k, sizeof(unsigned int));
    double startTime = clock();
    kernelFindKey <<< Nblocks, Nthreads >>> (g, p, h, d_k);
    hipMemcpy(h_k, d_k, sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    x = *h_k;
    double endTime = clock();
    printf("Secret key found! x = %u \n", x);

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;
    printf("Searching all keys took %g seconds, throughput was %g values tested per second. \n", totalTime, throughput);
    }
  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
    unsigned int bufferSize = 1024;
    unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
    unsigned int charsPerInt = (n-1)/8;
    unsigned int Nchars = charsPerInt*Nints;

    double decryptStart = clock();
    ElGamalDecrypt(Zmessage, a, Nints, p, x);
    convertZToString(Zmessage, Nints, message, Nchars);
    double decryptEnd = clock();

    double decryptTime = (decryptEnd = decryptStart)/CLOCKS_PER_SEC;
    printf("decrypted message = \"%s\"\n", message);
    printf("It took %g seconds to decrypt the message. \n", decryptTime);
    printf("\n");
    
    free(Zmessage);
    free(h_k);
    hipFree(d_k);
  return 0;
}
  


